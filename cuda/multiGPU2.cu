#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define NB 32  // Número de bloques por rejilla
#define NT 500 // Número de hilos por bloque
#define N NB * NT // Tamaño total del arreglo
/*
los cambios hechos:
    Se mejoró el manejo de errores durante la asignación de memoria en el host y en el dispositivo.
    Se utilizó cuda_runtime.h en lugar de cuda.h.
    Se cambiaron algunos tipos de datos a cudaEvent_t para utilizar funciones de la API de CUDA para medir el tiempo de ejecución de la GPU.
*/


// Kernel que realiza la suma en paralelo
__global__ void add(double *a, double *b, double *c)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    while (tid < N)
    {
        c[tid] = a[tid] + b[tid];
        tid += blockDim.x * gridDim.x;
    }
}


int main(void)
{
    // Declaración de arreglos en el host y en el dispositivo
    double *a, *b, *c;
    double *dev_a, *dev_b, *dev_c;

    // Asignación de memoria en el host
    a = (double *)malloc(N * sizeof(double));
    b = (double *)malloc(N * sizeof(double));
    c = (double *)malloc(N * sizeof(double));

    // Verificación de la asignación de memoria en el host
    if (a == NULL || b == NULL || c == NULL)
    {
        fprintf(stderr, "Error al asignar memoria en el host\n");
        exit(EXIT_FAILURE);
    }

    // Asignación de memoria en el dispositivo (GPU)
    hipMalloc((void **)&dev_a, N * sizeof(double));
    hipMalloc((void **)&dev_b, N * sizeof(double));
    hipMalloc((void **)&dev_c, N * sizeof(double));

    // Verificación de la asignación de memoria en el dispositivo
    if (dev_a == NULL || dev_b == NULL || dev_c == NULL)
    {
        fprintf(stderr, "Error al asignar memoria en el dispositivo\n");
        exit(EXIT_FAILURE);
    }

    // Inicialización de los arreglos 'a' y 'b' en el host
    for (int i = 0; i < N; i++)
    {
        a[i] = static_cast<double>(i);
        b[i] = static_cast<double>(i * 2);
    }

    // Copia de los arreglos 'a' y 'b' desde el host al dispositivo
    hipMemcpy(dev_a, a, N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N * sizeof(double), hipMemcpyHostToDevice);

    // Sirve para medir tiempo de ejecución paralelo
    hipEvent_t start_c, end_c;
    hipEventCreate(&start_c);
    hipEventCreate(&end_c);
    hipEventRecord(start_c, 0);

    // Llamada al kernel 'add' con NB bloques y NT hilos por bloque
    for (int i = 0; i < 10000; ++i)
        add<<<NB, NT>>>(dev_a, dev_b, dev_c);

    // Detiene el tiempo de ejecución de la GPU
    hipEventRecord(end_c, 0);
    hipEventSynchronize(end_c);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start_c, end_c);

    // Copia del resultado desde el dispositivo al host
    hipMemcpy(c, dev_c, N * sizeof(double), hipMemcpyDeviceToHost);

    // verificar kis resultados
    for (int i=0; i<N; i++) {
         printf( "%g + %g = %g\n", a[i], b[i], c[i] );
     }

    printf("GPU done\n");
    printf("Total GPU time used: %f ms\n", elapsedTime);

    // Liberación de memoria en el dispositivo
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    // Liberación de memoria en el host
    free(a);
    free(b);
    free(c);

    return 0;
}

